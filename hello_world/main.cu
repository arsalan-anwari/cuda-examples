
#include <hip/hip_runtime.h>
#include <stdio.h>

// This kernel runs on the GPU and prints the thread's identifiers
__global__ void kernel() {
  printf("Hello from block %d thread %d\n", blockIdx.x, threadIdx.x);
}

int main() {
  // Launch the kernel on the GPU with four blocks of six threads each
  kernel<<<4,6>>>();

  // Check for CUDA errors
  if(hipDeviceSynchronize() != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(hipPeekAtLastError()));
  }
  return 0;
}